/*
 * Created by Harshavardhan Patil on 9/30/16.
 *
 *
 *  Matrix Normalization using CUDA :
 *  - The generated input values are stored by inverting the matrix. i.e All the attributes of a column which needs to be normalized
 *    are stored as elements of a row. So that while normalizing the threads in a block will access nearby elements which will
 *    optimize the code
 *  - The number of threads is hard coded to 16. This value is finalized after multiple iterations with values (32, 64, 128, 256, 512).
 *    This value is used along with the size of the matrix to calculate number of blocks.
 *  - Each block is assigned to calculate sum and squares of one row.
 *    In a block each thread will read one element and put it in a shared memory. Once this is done, partial sum is calculated and
 *    each block stores it's partial sum in the global memory area using it's block Id as index.
 *  - Once partial sum calculation is done by all the blocks, another kernal function is launched with this partial
 *    sum as the input in a single block. This block calculates the final sum and squares.
 *  - Partial sum using reduction works only if the number of elements passed to the block is a power of 2.
 *    So to avoid wrong calculation when number of blocks is not power of 2 (in calculateFinalSum method).
 *    The input array argument lentgh is set to the nearest power of 2 for the number of blocks and value 0 is set to those indices
 *    which are greater than number of blocks.
 *  - Then the population standard deviation is calculated for that row using formula (sumOfSquares + N * powf(mean, 2.0) - 2 * mean * sumOfTheElements)/N;
 *    Where,
 *        N - Size of the Matrix
 *  - The above values are used to calculate standard score of each element in that row.
 *  - The computed values are stored in the output matrix at their inverse position. This operation is done for all the elements
 *
 *  Steps to compile and execute
    -----------------------------
    1) Go to folder "/home/hpatil2/hw4"
    2) run : qlogin -q interactive.q
    3) run : cd /home/hpatil2/hw4
    4) run : nvcc MatrixNormalizationCuda.cu -o MatrixNormalizationCuda
    5) run : ./MatrixNormalizationCuda 15000 4

    In step 5 : [argument1 (15000) is MATRIX_SIZE, this is mandatory to pass. and maximum value it can take is 15000]
                [argument2 (4) is seed value, this is an optional field]
 *
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>

/* Program Parameters */
#define MAXN 15000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
volatile float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

/* returns a seed for srand based on the time */
unsigned int time_seed() {
    struct timeval t;
    struct timezone tzdummy;

    gettimeofday(&t, &tzdummy);
    return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
    int seed = 0;  /* Random seed */
    char uid[32]; /*User name */

    /* Read command-line arguments */
    srand(time_seed());  /* Randomize */

    if (argc == 3) {
        seed = atoi(argv[2]);
        srand(seed);
        printf("Random seed = %i\n", seed);
    }
    if (argc >= 2) {
        N = atoi(argv[1]);
        if (N < 1 || N > MAXN) {
            printf("N = %i is out of range.\n", N);
            exit(0);
        }
    }
    else {
        printf("Usage: %s <matrix_dimension> [random seed]\n",
               argv[0]);
        exit(0);
    }

    /* Print parameters */
    printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
    int row, col;

    printf("\nInitializing...\n");
    for (row = 0; row < N; row++) {
        for (col = 0; col < N; col++) {
            A[row][col] = (float)rand() / 32768.0;
            B[row][col] = 0.0;
        }
    }

}

/* Print input matrices */
void print_inputs() {
    int row, col;

    if (N < 10) {
        printf("\nA =\n\t");
        for (col = 0; col < N; col++) {
            for (row = 0; row < N; row++) {
                printf("%5.2f%s", A[row][col], (row < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

void print_B() {
    int row, col;

    if (N < 10) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row][col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

/*
 *  This method calculates sum and square of given block
 */
__global__ void calculateBlockSum(const float *input, float *sumResults, float *squareResults, const size_t n)
{
    __shared__ float smSum[512];
    __shared__ float smSquare[512];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    float x = 0;

    if(i < n) {
        x = input[i];
    }
    smSum[tx] = x;
    smSquare[tx] = x*x;
    __syncthreads();

    int j;
    for(j = blockDim.x / 2; j > 0; j >>= 1) {

        if(tx < j) {
            smSum[tx] = smSum[tx]+ smSum[tx + j];
            smSquare[tx] = smSquare[tx] + smSquare[tx + j];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {

        sumResults[blockIdx.x] = smSum[0];
        squareResults[blockIdx.x] = smSquare[0];
    }
}

/*
 *  This method calculates final sum and square
 */
__global__ void calculateFinalSum(float *sumResults, float *squareResults, const size_t size, float *finalSumResult, float *finalSigmaResult)  {

    __shared__ float smSum[512];
    __shared__ float smSquare[512];
    int tx = threadIdx.x;

    if(tx < size) {
        smSum[tx] = sumResults[tx];
        smSquare[tx] = squareResults[tx];
    }
    __syncthreads();

    int i;
    for(i = size/2; i > 0; i >>= 1) {

        if(tx < i) {
            smSum[tx] = smSum[tx] + smSum[tx + i];
            smSquare[tx] = smSquare[tx] + smSquare[tx + i];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {

        finalSumResult[0] = smSum[0];
        finalSigmaResult[0] = smSquare[0];
    }
}

float* calculateSum (float *input, size_t n, float *dMatrixA, float *sumValue, float *sigmaValue, size_t blockSize, size_t totalBlocks, int nextNearestPowerOf2, float *finalSumResult, float *finalSigmaResult) {

    float *results = (float *)malloc(sizeof(float) * 2);

    hipMemcpy(dMatrixA, input, sizeof(float) * N, hipMemcpyHostToDevice);
    calculateBlockSum<<<totalBlocks, blockSize>>> (dMatrixA, sumValue, sigmaValue, n);
    calculateFinalSum<<<1,totalBlocks>>>(sumValue, sigmaValue, nextNearestPowerOf2, finalSumResult, finalSigmaResult);

    hipMemcpy(&results[0], &finalSumResult[0], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&results[1], &finalSigmaResult[0], sizeof(float), hipMemcpyDeviceToHost);

    return results;
}

void matrixNorm() {

    int row, column;
    float mu, sigma;
    float *sumValue = 0, *sigmaValue = 0, *dMatrixA = 0;
    float *finalSumResult = 0, *finalSigmaResult = 0;


    printf("Parallel Computing.\n");

    size_t blockSize = 16;
    size_t totalBlocks;

    if(N%blockSize == 0){
        totalBlocks = (N/blockSize);
    } else {
        totalBlocks = (N/blockSize) + 1;
    }

    int nextNearestPowerOf2 = pow(2, ceil(log(totalBlocks)/log(2)));

    hipMalloc((void**)&sumValue, sizeof(float) * (nextNearestPowerOf2));
    hipMemset(sumValue, 0.0, sizeof(float) * nextNearestPowerOf2);
    hipMalloc((void**)&sigmaValue, sizeof(float) * (nextNearestPowerOf2));
    hipMemset(sigmaValue, 0.0, sizeof(float) * nextNearestPowerOf2);
    hipMalloc((void**)&dMatrixA, sizeof(float) * N);
    hipMalloc((void**)&finalSumResult, sizeof(float));
    hipMalloc((void**)&finalSigmaResult, sizeof(float));

    for (column=0; column < N; column++) {

        mu = 0.0;
        float *result;
        result = calculateSum ((float *)A[column], N, dMatrixA, sumValue, sigmaValue, blockSize, totalBlocks, nextNearestPowerOf2,finalSumResult,finalSigmaResult);
        mu = result[0] / (float) N;
        sigma = (result[1] + N * powf(mu, 2.0) - 2 * mu * result[0])/(float)N;

        for (row=0; row < N; row++) {
            if (sigma == 0.0) {
                B[row][column] = 0.0;
            } else {
                B[row][column] = (A[column][row] - mu) / sigma;
            }
        }
    }
    hipFree(sumValue);
    hipFree(sigmaValue);
    hipFree(dMatrixA);
}


int main(int argc, char **argv) {

    struct timeval etstart, etstop;
    struct timezone tzdummy;
    clock_t etstart2, etstop2;
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;

    parameters(argc, argv);
    initialize_inputs();
    print_inputs();

    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    etstart2 = times(&cputstart);

    matrixNorm();

    gettimeofday(&etstop, &tzdummy);
    etstop2 = times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    print_B();

    printf("\nElapsed time = %g ms.\n",
           (float)(usecstop - usecstart)/(float)1000);

    printf("(CPU times are accurate to the nearest %g ms)\n",
           1.0/(float)CLOCKS_PER_SEC * 1000.0);
    printf("My total CPU time for parent = %g ms.\n",
           (float)( (cputstop.tms_utime + cputstop.tms_stime) -
                    (cputstart.tms_utime + cputstart.tms_stime) ) /
           (float)CLOCKS_PER_SEC * 1000);
    printf("My system CPU time for parent = %g ms.\n",
           (float)(cputstop.tms_stime - cputstart.tms_stime) /
           (float)CLOCKS_PER_SEC * 1000);
    printf("My total CPU time for child processes = %g ms.\n",
           (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
                    (cputstart.tms_cutime + cputstart.tms_cstime) ) /
           (float)CLOCKS_PER_SEC * 1000);
    printf("--------------------------------------------\n");

    exit(0);
}